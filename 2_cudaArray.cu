
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

#define ROW 3
#define COL 4

__global__ void collapse(int *a, int *c) {

	int total = 0;
	for(int i = 0; i < COL; i++)
	{
		total = total + a[blockIdx.x*COL + i];
	}
	c[blockIdx.x] = total;
}

int main(void)
{
	int array[ROW][COL];
	int c[ROW];
	int *dev_a; 
	int *dev_c; 

	int size_2D = ROW*COL*sizeof(int);
	int size_c = ROW*sizeof(int);
	
	hipMalloc((void**)&dev_a, size_2D);
	hipMalloc((void**)&dev_c, size_c);

	
	for (int i = 0; i < ROW; i++)
	{
		if(i == ROW -1)
		{
			for(int j = 0; j < COL; j++)
			{
				array[i][j] = (j*2);
				printf("%i ", array[i][j]);
			}
		}
		else
		{
			for(int j = 0; j < COL; j++)
			{
				array[i][j] = j;
				printf("%i ", array[i][j]);
			}
		}
		printf("\n");
	}

	hipMemcpy(dev_a, array, size_2D, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, size_c,	 hipMemcpyHostToDevice);
	
	collapse<<< ROW, COL >>> (dev_a, dev_c);	

	hipMemcpy(c, dev_c, size_c, hipMemcpyDeviceToHost);	

	printf("\n");
	for(int i = 0; i < ROW; i++)
	{
	    printf("%i\n", c[i]);
	}	

	
	hipFree (dev_a);
	hipFree (dev_c);

}
	
