#include "hip/hip_runtime.h"
// This is a simple ray tracer that shoots rays top down toward randomly
// generates spheres and draws the sphere in a random color based on where
// the ray hits it.

#include "FreeImage.h"
#include "stdio.h"

#define DIM 1028
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
    // Tells us if a ray hits the sphere; return the
    // depth of the hit, or -infinity if the ray misses the sphere
    __device__ float hit( float ox, float oy, float *n ) 
    {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius)
        {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};

#define SPHERES 40

// Loops through each pixel in the image (represented by arrays of
// red, green, and blue) and then for each pixel checks if a ray from
// top down hits one of the randomly generated spheres.
// If so, calculate a shade of color based on where the ray hits it.
__global__ void drawSpheres(Sphere *spheres, char *red, char *green, char *blue)
{
    
 	int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.x + blockIdx.y * blockDim.y;
	
	float   ox = (x - DIM/2);
	float   oy = (y - DIM/2);

	float   r=0, g=0, b=0;
	float   maxz = -INF;
	for(int i=0; i<SPHERES; i++)
 	{
        	float   n;
        	float   t = spheres[i].hit( ox, oy, &n );
        	if (t > maxz)
		{
			// Scale RGB color based on z depth of sphere
            		float fscale = n;
            		r = spheres[i].r * fscale;
            		g = spheres[i].g * fscale;
			b = spheres[i].b * fscale;
            		maxz = t;
        	} 
        }
	
    	int offset = x + y * blockDim.x * gridDim.x;
    	red[offset] = (char) (r * 255);
    	green[offset] = (char) (g * 255);
    	blue[offset] = (char) (b * 255);
    
}

int main()
{
  FreeImage_Initialise();
  atexit(FreeImage_DeInitialise);
  FIBITMAP * bitmap = FreeImage_Allocate(DIM, DIM, 24);
  srand(time(NULL));

 
  char red[DIM][DIM];
  char green[DIM][DIM];
  char blue[DIM][DIM];
  char *dev_red; 
  char *dev_green; 
  char *dev_blue;
  Sphere *dev_spheres;
  
 
 
  // Dynamically create enough memory for DIM * DIM array of char.
  // By making these dynamic rather than auto (e.g. char red[DIM][DIM])
  // we can make them much bigger since they are allocated off the heap
  hipMalloc((void**)&dev_red, DIM*DIM*sizeof(char));
  hipMalloc((void**)&dev_green, DIM*DIM*sizeof(char));
  hipMalloc((void**)&dev_blue, DIM*DIM*sizeof(char));
  hipMalloc((void**)&dev_spheres, sizeof(Sphere)*SPHERES);

  

  //Create random spheres at different coordinates, colors, radius
  //Sphere spheres[SPHERES];
 
  Sphere *spheres =(Sphere*)malloc(sizeof(Sphere)*SPHERES); 
  for (int i = 0; i<SPHERES; i++)
  {
        spheres[i].r = rnd( 1.0f );
        spheres[i].g = rnd( 1.0f );
        spheres[i].b = rnd( 1.0f );
        spheres[i].x = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].y = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].z = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].radius = rnd( 100.0f ) + 20;
  } 

  hipMemcpy(dev_spheres, spheres, sizeof(Sphere)*SPHERES, hipMemcpyHostToDevice);
  free(spheres);

  dim3 grid(DIM/16,DIM/16);
  dim3 threads(16,16);
  drawSpheres<<<grid,threads>>>(dev_spheres, dev_red, dev_green, dev_blue);
 
  hipMemcpy(dev_red, red, DIM*DIM*sizeof(char), hipMemcpyDeviceToHost);
  hipMemcpy(dev_green, green, DIM*DIM*sizeof(char), hipMemcpyDeviceToHost);
  hipMemcpy(dev_blue, blue, DIM*DIM*sizeof(char), hipMemcpyDeviceToHost);
 

  RGBQUAD color;
  for (int i = 0; i < DIM; i++)
  {
    for (int j = 0; j < DIM; j++)
    {
      //int index = j*DIM + i;
      color.rgbRed = red[i][j];
      color.rgbGreen = green[i][j];
      color.rgbBlue = blue[i][j];
      FreeImage_SetPixelColor(bitmap, i, j, &color);
    }
  }
  

  FreeImage_Save(FIF_PNG, bitmap, "ray.png", 0);
  FreeImage_Unload(bitmap);
  hipFree(dev_red);
  hipFree(dev_green);
  hipFree(dev_blue);
  hipFree(dev_spheres);

  return 0;
}
